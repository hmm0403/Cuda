
#include <hip/hip_runtime.h>
#include <cstdio>
#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_),                                 \
              hipGetErrorString(status_));                              \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define MAX_GPUS 8 // maximum number of GPUs possible

int main() {
    int count;
    hipGetDeviceCount(&count);
    printf("Number of devices: %d\n",count);
    hipDeviceProp_t props[MAX_GPUS];
    for(int d = 0; d< count; ++d){
        printf("device %d:\n", d);
        hipGetDeviceProperties(&props[d], d);

        printf("\t\tname : %s\n", props[d].name);
        printf("\t\tmultiProcessorCount : %d\n", props[d].multiProcessorCount);
        printf("\t\tmaxThreadsPerBlock : %d\n", props[d].maxThreadsPerBlock);
        printf("\t\ttotalGlobalMem : %lu\n", props[d].totalGlobalMem);
        printf("\t\tsharedMemPerBlock : %lu\n", props[d].sharedMemPerBlock);
  }
  return 0;
}
