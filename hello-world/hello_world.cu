
#include <hip/hip_runtime.h>
#include <cstdio>

// CUDA 커널을 정의해 GPU에서 실행할 코드 구현
__global__ void hello_world() {
  printf("Thread %d: Hello, World!\n", threadIdx.x);
}

int main() {
    hello_world<<<1, 32>>>();
    hipDeviceSynchronize();
  return 0;
}
