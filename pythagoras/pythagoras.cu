
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *pd){

    int a = *pa;
    int b = *pb;
    int c = *pc;

    if((a * a + b * b) == c * c ){
        *pd = 1;
    }else{
        *pd = 0;
    }

}

int main(int argc, char *argv[]) {
    if(argc != 4){
        printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
        return 0;
    }

    int *d_a, *d_b, *d_c, *d_d;
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));
    hipMalloc(&d_d, sizeof(int));

    int a = atoi(argv[1]);
    int b = atoi(argv[2]);
    int c = atoi(argv[3]);
    int d;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice);
    pythagoras<<<1, 10>>>(d_a, d_b, d_c, d_d);
    hipMemcpy(&d, d_d, sizeof(int), hipMemcpyDeviceToHost);
    if(d == 1){
        printf("YES\n");
    }
    else{
        printf("No\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    return 0;
}
